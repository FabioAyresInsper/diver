#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"
#include "cutil_math.h"

#define NUM_THREADS 384
#define NUM_BLOCKS 1024
#define MAX_HIT 8

#define PARAM_SIZE 3168

__device__ __forceinline__ float sigmoid (float x)
{
    return 1.0 / (1.0 + __expf (-x));
}


static texture <int, 3, hipReadModeElementType> c_voxel_map;
__device__ __constant__ float params_cache[PARAM_SIZE];

/**
 * Same as the mlp evaluation code in diver-rt repo.
 */
__global__ void mlp_eval_kernel(
            int batch_size,
            const float *__restrict__ coord,
            const float *__restrict__ voxels,
            const int *__restrict__ voxel_map,
            const float *__restrict__ v,
            float *__restrict__ rgba,
            bool* __restrict__ mask
        ) {
    
    constexpr float frequency_bands[4] 
        = {3.141592653589793f,6.283185307179586f,12.566370614359172f,25.132741228718345f};
    
    int b_idx = blockIdx.x;
    int tx = ((b_idx%50)*blockDim.x) + threadIdx.x;
    int ty = ((b_idx/50)*blockDim.y) + threadIdx.y;

    int batch_index = 800*ty + tx; 
    
    
    while (batch_index < batch_size) {
        if (mask[batch_index]) {
            b_idx += NUM_BLOCKS;
            tx = ((b_idx%50)*blockDim.x) + threadIdx.x;
            ty = ((b_idx/50)*blockDim.y) + threadIdx.y;
            batch_index = 800*ty + tx;
            continue;
        }
        for (int hit=0; hit < MAX_HIT; hit++) {
            int mask_idx = batch_index*6 + hit*batch_size*6;
    
            float x0 = coord[mask_idx];
            float y0 = coord[mask_idx+1];
            float z0 = coord[mask_idx+2];
            float x1 = coord[mask_idx+3];
            float y1 = coord[mask_idx+4];
            float z1 = coord[mask_idx+5];
            if (x0 < 0) {
                mask[batch_index] = true;
                break;
            }
    
            int cx = min(int(x0+1e-4f),int(x1+1e-4f));
            int cy = min(int(y0+1e-4f),int(y1+1e-4f));
            int cz = min(int(z0+1e-4f),int(z1+1e-4f));
            x0 -= cx;
            x1 -= cx;
            y0 -= cy;
            y1 -= cy;
            z0 -= cz;
            z1 -= cz;
            float z01 = z0 + z1;
            float y01 = y0 + y1;
            float x01 = x0 + x1;
            float w[8];
            w[7]=(2*x0*y0*z0+2*x1*y1*z1+x01*y01*z01)/12;
            float w2=(y01*z01+y0*z0+y1*z1)/6;
            w[5]=(x01*z01+x0*z0+x1*z1)/6-w[7];
            float w4=(x01*y01+x0*y0+x1*y1)/6;
    
            x01 *= 0.5f;
            y01 *= 0.5f;
            z01 *= 0.5f;
    
            w[4] = z01-w[5]-w2;
            w[0] = 1-w[4]-y01-x01+w4;
            w[6] = w2-w[7];
            w[2] = y01-w[6]-w4;
            w[1] = x01 -w[5] - w4;
            w[3] = -w[7] +w4;
    
            float buffer1[32];
            int param_offset = 0;
            // load features
            #pragma unroll
            for (int i = 0; i < 32; i++) {
                buffer1[i] = params_cache[param_offset];
                param_offset += 1;
            }

            #pragma unroll
            for (int j = 0; j < 8; j++) {
                int dx = j%2+cx;
                int dy = (j/2)%2+cy;
                int dz = j/4+cz;
                
                int v_idx = tex3D(c_voxel_map,dx,dy,dz);
                #pragma unroll
                for (int i = 0; i < 32; i++) {
                    buffer1[i] += w[j]*voxels[v_idx*32+i];
                }
            }
    
            // first mlp
            float buffer2[32];
            #pragma unroll
            for (int i=0; i < 32; i++) {
                buffer2[i] = params_cache[param_offset];
                param_offset += 1;
            }
    
            #pragma unroll
            for (int i=0; i < 32; i++) {
                #pragma unroll
                for (int j=0; j < 32; j++) {
                    buffer2[j] += params_cache[param_offset]*fmaxf(buffer1[i],0.0f);
                    param_offset += 1;
                }
            }
    
            // density mlp
            float sigma = params_cache[param_offset];
            param_offset += 1;
            #pragma unroll
            for (int i = 0; i < 32; i++) {
                sigma += params_cache[param_offset]*fmaxf(buffer2[i],0.0f);
                param_offset += 1;
            }
            sigma = -fmaxf(sigma,0.0f);
            sigma = 1.0f - __expf(sigma);
            
            if (sigma < 1e-2f) {
                continue;
            }
    
            // second mlp
            float buffer3[32];
            #pragma unroll
            for (int i =0; i < 32; i++) {
                buffer3[i] = params_cache[param_offset];
                param_offset += 1;
            }
            #pragma unroll
            for (int i =0; i < 32; i++) {
                #pragma unroll
                for (int j =0; j < 32; j++) {
                    buffer3[j] += params_cache[param_offset]*fmaxf(buffer2[i],0.0f);
                    param_offset += 1;
                }
            }
            // cat view dependent
            #pragma unroll
            for (int i = 0; i < 3; i++) {
                float input_elem = v[batch_index*3+i];
                #pragma unroll
                for (int e =0; e <9; e++) {
                    float embedded_input_elem;
                    if (e == 0) {
                        embedded_input_elem = input_elem;
                    } else if (e < 5) {
                        embedded_input_elem = __sinf(frequency_bands[e-1]*input_elem);
                    } else {
                        embedded_input_elem = __cosf(frequency_bands[e-5]*input_elem);
                    }
    
                    #pragma unroll
                    for (int j = 0; j < 32; j++) {
                        buffer3[j] += params_cache[param_offset]*embedded_input_elem;
                        param_offset += 1;
                    }
                }
            }
    
            // last mlp
            float buffer4[3];
            #pragma unroll
            for (int i=0; i < 3; i++ ) {
                buffer4[i] = params_cache[param_offset];
                param_offset += 1;
            }
            #pragma unroll
            for (int i = 0; i < 32; i++) {
                #pragma unroll
                for (int j =0; j < 3; j++) {
                    buffer4[j] += params_cache[param_offset]*fmaxf(buffer3[i],0.0f);
                    param_offset += 1;
                }
            }
    
            float acc_sigma = rgba[batch_index*4+3];
            float new_sigma = acc_sigma*sigma;
            #pragma unroll
            for (int i = 0; i < 3; i++) {
                rgba[batch_index*4+i] += new_sigma*sigmoid(buffer4[i]);
            }
            new_sigma = acc_sigma*(1-sigma);
            rgba[batch_index*4+3] = new_sigma;
            if (new_sigma < 1e-2f) {
                mask[batch_index] = true;
                break;
            }
        }

        b_idx += NUM_BLOCKS;
        tx = ((b_idx%50)*blockDim.x) + threadIdx.x;
        ty = ((b_idx/50)*blockDim.y) + threadIdx.y;
        batch_index = 800*ty + tx;
    }
}


void mlp_eval_wrapper(
  int batch_size,
  const float* coord,
  const float* voxels,
  const int* voxel_map,
  const float* v,
  float* rgba,
  bool* mask
){

  dim3 block(16,24);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  mlp_eval_kernel<<<NUM_BLOCKS, block,0,stream>>>(
      batch_size,
      coord,
      voxels,
      voxel_map,
      v,
      rgba,mask);
  
  CUDA_CHECK_ERRORS();
  hipDeviceSynchronize();
}



/**
* Same as the upload weight code in diver-rt repo
*/
void upload_weight_wrapper(
    int device_id,
    int chunk_num, int chunk_scale, int chunk_size,
    const float* params,
    const float* voxel_chunk,
    const int* chunk_map
) {
    hipSetDevice(device_id);



    // allocate map
    hipChannelFormatDesc cf = hipCreateChannelDesc<int>();
    hipArray * voxel_map_array = 0;
    hipMalloc3DArray(&voxel_map_array,&cf,make_hipExtent(chunk_scale,chunk_scale,chunk_scale),0);
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr((void*)chunk_map,chunk_scale*sizeof(int),chunk_scale,chunk_scale);
    copyParams.dstArray = voxel_map_array;
    copyParams.extent = make_hipExtent(chunk_scale,chunk_scale,chunk_scale);
    copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);
    
    c_voxel_map.normalized = 0;
    c_voxel_map.filterMode = hipFilterModePoint;
    c_voxel_map.addressMode[0] = hipAddressModeClamp;
    c_voxel_map.addressMode[1] = hipAddressModeClamp;
    c_voxel_map.addressMode[2] = hipAddressModeClamp;
    hipBindTextureToArray(c_voxel_map,voxel_map_array,cf); 
    

    hipMemcpyToSymbol(HIP_SYMBOL(params_cache),params,PARAM_SIZE*sizeof(float),0,hipMemcpyHostToDevice);
    hipDeviceSynchronize();
}

